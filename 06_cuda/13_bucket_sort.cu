
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void sort(int *key, int *bucket, int range, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
    __syncthreads();
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
    __syncthreads();
  }
  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
      __syncthreads();
    }
  }
}

int main() {
  int n = 50;
  int range = 5;
//  std::vector<int> key(n);
  int *key;
  hipMallocManaged(&key, n*sizeof(int));

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

//  std::vector<int> bucket(range);
  int *bucket; 
  hipMallocManaged(&bucket, range*sizeof(int));
  
//    for (int i=0; i<range; i++) {
//    bucket[i] = 0;
//  }
//  for (int i=0; i<n; i++) {
//    bucket[key[i]]++;
//  }
//  for (int i=0, j=0; i<range; i++) {
//    for (; bucket[i]>0; bucket[i]--) {
//      key[j++] = i;
//    }
//  }

  sort<<<1,n>>>(key, bucket, range, n);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(key);
  hipFree(bucket);
}
